#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAException.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CUDA_ERR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define WARP_SIZE 32
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val){
    #pragma unroll
    for(int stride = kWarpSize >> 1; stride >= 1; stride >>= 1){
        val += __shfl_xor_sync(0xffffffff, val, stride);
    }
    return val;
}

template<const int NUM_THREADS = 256>
__device__ __forceinline__ float block_reduce_sum_f32(float v){
    const int NUM_WARPS = NUM_THREADS / WARP_SIZE;
    static __shared__ float shared[NUM_WARPS];

    int lane = threadIdx.x % WARP_SIZE;
    int wid = threadIdx.x / WARP_SIZE;
    
    v = warp_reduce_sum_f32<WARP_SIZE>(v);
    if(lane == 0) shared[wid] = v;
    __syncthreads();
    v = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
    v = warp_reduce_sum_f32<NUM_WARPS>(v);
    return v;
}

template< const int NUM_THREADS=256>
__global__ void rmsnorm_f32_kernel(float* x, float* y,float* w, int N, int k){
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    int block_id = blockIdx.x;

    __shared__ float sum_sqx;
    float value = (global_tid < N * k) ? x[global_tid] : 0.0f;
    float sum = block_reduce_sum_f32<NUM_THREADS>(value * value);
    if (local_tid == 0) sum_sqx = sum;
    __syncthreads();
    y[global_tid] = global_tid < N * k ? w[local_tid] * value / sqrtf(sum_sqx / k) : 0.0f;
}
torch::Tensor launch_rmsnorm_kernel_fp32(torch::Tensor x, torch::Tensor w) {
    CHECK_INPUT(x);    
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(x.device());
    torch::Tensor y = torch::empty_like(x, options);
    int batch_size = x.size(0);
    int seqlen = x.size(1);
    int hidden_size = x.size(2);
    int N = batch_size * seqlen;
    int K = hidden_size;
    rmsnorm_f32_kernel<768><<<N, 768>>>(x.data_ptr<float>(), y.data_ptr<float>(), w.data_ptr<float>(), N, K);
    return y;
}