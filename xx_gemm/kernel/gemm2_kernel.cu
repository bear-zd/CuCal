#include <hip/hip_runtime.h>
#include <stdio.h>
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

__global__ void naiveSgemm(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    if (m < M && n < N) {
        float psum = 0.0;
        #pragma unroll
        for (int k = 0; k < K; k++) {
            psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = psum;
    }
}

__global__ void shared_gemm(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {
        const int BM = 128;
        const int BN = 128;
        const int BK = 8;
        const int TM = 8;
        const int TN = 8;
        
        const int bx = blockIdx.x;
        const int by = blockIdx.y;
        const int tx = threadIdx.x;
        const int ty = threadIdx.y;
        const int tid = tx + ty * blockDim.x;

        __shared__ float s_a[BM][BK];
        __shared__ float s_b[BK][BN];

        float r_c[TM][TN] = {0.0};

        int load_a_smem_m = tid >> 1; 
        int load_a_smem_k = (tid & 1) << 2;
        /*  used in the following code:
            FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr]);
        */
        // pick 4 block each time. because of the BK = 8 and 4 block of float is satified the data storage in memory
        int load_b_smem_k = tid >> 5;
        int load_b_smem_n = (tid & 31) << 2;
        /*  used in the following code:
            FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr]);
        */
        // BK = 8, in the vertical , BK * 4 int data = 32. 

        int load_a_gmem_m = by * BM + load_a_smem_m ;
        int load_b_gmem_n = bx * BN + load_b_smem_n ;
        // only transfer the local smem index to global index (mentioned that this is only the m,n value)
        for(int bk = 0; bk < (K + BK - 1)/BK; bk++ ){
            int load_a_gmem_k = bk * BK + load_a_smem_k;
            int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
            FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr]); 
            int load_b_gmem_k = bk * BK + load_b_smem_k;
            int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
            FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr]);
            // copy the data from global to shared memory.
            __syncthreads();
            // split the code above and below. Now consider the data in shared memory;
            #pragma unroll
            for(int k=0; k < BK; k++){
                #pragma unroll
                for(int m = 0; m < TM; m++){
                    #pragma unroll
                    for(int n = 0; n < TN; n++){
                        int comp_a_smem_m = ty * TM + m;
                        int comp_b_smem_n = tx * TN + n;
                        r_c[m][n] += s_a[comp_a_smem_m][k] * s_b[k][comp_b_smem_n];
                    }
                }
            }
            // a bit too complex. Each time get a col of s_a and a row of s_b to multiple and add to the r_c;
            // after sum above. each thread store the r_c matrix in local but not write to c matrix;
            __syncthreads();
        }
        #pragma unroll
        for(int i=0; i < TM ; i+=1){
            int store_c_gmem_m = by * BM + ty * TM + i;
            #pragma unroll
            for(int j=0; j < TN; j+=4){
                int store_c_gmem_n = bx * BN + tx * TN + j;
                int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
                FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][j]);
            }
        }
        
}


__global__ void sgemm_V3(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 32;
    const int BN = 32;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[2][BK][BM];
    __shared__ float s_b[2][BK][BN];

    float r_load_a[4];
    float r_load_b[4];
    float r_comp_a[TM];
    float r_comp_b[TN];
    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    {
        int load_a_gmem_k = load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        s_a[0][load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[0][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[0][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[0][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[0][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);
    }

    for (int bk = 1; bk < (K + BK - 1) / BK; bk++) {

        int smem_sel = (bk - 1) & 1;
        int smem_sel_next = bk & 1;

        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        #pragma unroll
        for (int tk = 0; tk < BK; tk++) {
            FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2         ]);
            FLOAT4(r_comp_a[4]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2 + BM / 2]);
            FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2         ]);
            FLOAT4(r_comp_b[4]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2 + BN / 2]);

            #pragma unroll
            for (int tm = 0; tm < TM; tm++) {
                #pragma unroll
                for (int tn = 0; tn < TN; tn++) {
                    r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                }
            }
        }

        s_a[smem_sel_next][load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[smem_sel_next][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[smem_sel_next][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[smem_sel_next][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[smem_sel_next][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);

        __syncthreads();
    }

    #pragma unroll
    for (int tk = 0; tk < BK; tk++) {
        FLOAT4(r_comp_a[0]) = FLOAT4(s_a[1][tk][ty * TM / 2         ]);
        FLOAT4(r_comp_a[4]) = FLOAT4(s_a[1][tk][ty * TM / 2 + BM / 2]);
        FLOAT4(r_comp_b[0]) = FLOAT4(s_b[1][tk][tx * TN / 2         ]);
        FLOAT4(r_comp_b[4]) = FLOAT4(s_b[1][tk][tx * TN / 2 + BN / 2]);

        #pragma unroll
        for (int tm = 0; tm < TM; tm++) {
            #pragma unroll
            for (int tn = 0; tn < TN; tn++) {
                r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
            }
        }
    }

    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
        FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
    }
    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + BM / 2 + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
        FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
    }
}

void launch_gemm2(float* a, float* b, float* c, int M, int N, int K) {
    // Define block and grid sizes

    int BM = 32;
    int BN = 32;
    dim3 blockSize(BN, BM); // 16x16 threads per block
    dim3 gridSize((N + BN - 1) / BN, (M + BM - 1) / BM);

    // Launch the kernel
    naiveSgemm<<<gridSize, blockSize>>>(a, b, c, M, N, K);
    hipDeviceSynchronize();
    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    // printf("%d", err);

}