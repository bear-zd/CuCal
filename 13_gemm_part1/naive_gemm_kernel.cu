#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAException.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CUDA_ERR(ans) { gpuAssert((ans), __FILE__, __LINE__); }

#define TILE_SIZE 32
#define BLOCK_SIZE 32

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void gemm_naive(float* x, float* y, float* out, int m, int n, int k){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < m && col < n){
        float sum = 0.0f;
        for(int i = 0; i < k; i++){
            sum += x[row * k + i] * y[i * n + col];
        }
        out[row * n + col] = sum;
    }
}

__global__ void gemm_shared(float* x, float* y, float* out, int M, int N, int K){
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float shared_x[TILE_SIZE][TILE_SIZE];
    __shared__ float shared_y[TILE_SIZE][TILE_SIZE];
    unsigned int length_bk = (K + TILE_SIZE - 1) / TILE_SIZE;
    float sum = 0.0f;
    for(int sblock_idx = 0; sblock_idx < length_bk; sblock_idx++){
        // load data into shared memory
        shared_x[ty][tx] = (row < M && (sblock_idx * TILE_SIZE + tx) < K) ? x[row * K + sblock_idx * TILE_SIZE + tx]:0.0f;
        shared_y[ty][tx] = (col < N && (sblock_idx * TILE_SIZE + ty) < K) ? y[(sblock_idx * TILE_SIZE + ty) * N + col]:0.0f;
        __syncthreads();
        #pragma unroll
        for(int i = 0; i < TILE_SIZE; i++){
            sum += shared_x[ty][i] * shared_y[i][tx];
        }
        __syncthreads();
    }
    if(row < M && col < N) out[row * N + col] = sum;
}



torch::Tensor gemm(torch::Tensor x, torch::Tensor y){
    CHECK_INPUT(x);
    CHECK_INPUT(y);
    int m = x.size(0);
    int n = y.size(1);
    int k = x.size(1);
    int grid_x = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int grid_y = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 grid(grid_x, grid_y);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    torch::Tensor z = torch::empty({m, n}, x.options());
    float* x_ptr = x.data_ptr<float>();
    float* y_ptr = y.data_ptr<float>();
    float* z_ptr = z.data_ptr<float>();
    gemm_shared<<<grid, block>>>(x_ptr, y_ptr, z_ptr, m, n, k);
    return z;
}