#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAException.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CUDA_ERR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define WARP_SIZE 32
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct __align__(8) MD
{
    float m;
    float d;
}; 


template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ MD warp_reduce_md_op(MD value) {
    unsigned int mask = 0xffffffff;
    #pragma unroll
    for(int stride = kWarpSize >> 1; stride >= 1; stride >>= 1) {
        MD other;
        other.m = __shfl_xor_sync(mask, value.m, stride);
        other.d = __shfl_xor_sync(mask, value.d, stride);

        bool value_bigger = (value.m > other.m);
        MD bigger_m = value_bigger ? value : other;
        MD smaller_m = value_bigger ? other : value;
        
        value.d = bigger_m.d + smaller_m.d * __expf(smaller_m.m - bigger_m.m);
        value.m = bigger_m.m;
    }
    return value;
}


template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_shffl_sum(float val){
    #pragma unroll
    for(int stride = kWarpSize >> 1; stride >= 1; stride >>= 1){
        val += __shfl_xor_sync(0xffffffff, val, stride);
    }
    return val;
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_max_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val = fmaxf(val, __shfl_xor_sync(0xffffffff, val, mask));
  }
  return val;
}

template<const int NUM_THREADS = 256>
__device__ float block_reduce_sum_f32(float val){
    constexpr int NUM_WARPS = (NUM_THREADS - 1 + WARP_SIZE)/ WARP_SIZE;
    int lane = threadIdx.x % WARP_SIZE;
    int warp = threadIdx.x / WARP_SIZE;
    static __shared__ float shared[NUM_WARPS];
    
    float t_val = warp_shffl_sum<WARP_SIZE>(val);
    if(lane == 0) shared[warp] = t_val;
    __syncthreads();

    t_val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
    t_val = warp_shffl_sum<NUM_WARPS>(t_val);
    t_val = __shfl_sync(0xffffffff, t_val, 0, 32);
    return t_val;
}


template<const int NUM_THREADS=256>
__device__ float block_reduce_max_f32(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  float value = warp_reduce_max_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = value;
  __syncthreads();
  value = (lane < NUM_WARPS) ? shared[lane] : -FLT_MAX;
  value = warp_reduce_max_f32<NUM_WARPS>(value);
  // WRAN: need to broadcast value to all threads within warp
  value = __shfl_sync(0xffffffff, value, 0, 32);
  return value;
}



template<const int NUM_THREADS = 256>
__global__ void safe_softmax_kernel(const float* x, float* y, float* total,int length){

    int local_tid = threadIdx.x;
    int global_tid = blockIdx.x * NUM_THREADS + threadIdx.x;
    float val = global_tid < length ? x[global_tid] : -FLT_MAX;
    float max_val = block_reduce_max_f32<NUM_THREADS>(val);
    float exp_val = global_tid < length ? expf(x[global_tid] - max_val) : 0.0f;
    float exp_sum = block_reduce_sum_f32<NUM_THREADS>(exp_val);
    if (local_tid == 0) {
      atomicAdd(total, exp_sum);
    }
    __threadfence(); 
    
    if (global_tid < length) y[global_tid] = exp_val / (*total); 
}


template<const int NUM_THREADS = 256 >
__global__ void online_softmax_kernel(const float* x, float* y, int length) {
  
    int local_tid = threadIdx.x;
    int global_tid = blockIdx.x * NUM_THREADS + threadIdx.x;
    const int WAPR_NUM = NUM_THREADS / WARP_SIZE;
    int warp_id = local_tid / WARP_SIZE;
    int lane_id = local_tid % WARP_SIZE;
    MD val;
    val.m = global_tid < length ? x[global_tid] : -FLT_MAX;
    val.d = global_tid < length ? 1.0f : 0.0f;

    __shared__ MD shared[ WAPR_NUM ]; 
    MD res = warp_reduce_md_op<WARP_SIZE>(val);

    if (lane_id == 0) shared[warp_id] = res; 
    __syncthreads();

    if (local_tid < WARP_SIZE) {
        MD block_res = shared[local_tid];
        block_res = warp_reduce_md_op<WAPR_NUM>(block_res); 
        if (local_tid == 0) {
            shared[0] = block_res; 
        }
    }
    __syncthreads();

    MD final_res = shared[0];
    float d_total_inverse = __fdividef(1.0f, final_res.d);
    if (global_tid < length) {
        y[global_tid] = __expf(x[global_tid] - final_res.m) * d_total_inverse;
    }
}



torch::Tensor launch_softmax_kernel_fp32(torch::Tensor x){
    CHECK_INPUT(x);    
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(x.device());
    const int N = x.numel();

    auto output = torch::zeros({N}, options);
    auto total = torch::zeros({1}, options);
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    safe_softmax_kernel<256><<<grid, block>>>(x.data_ptr<float>(), output.data_ptr<float>(), total.data_ptr<float>(), N);
    CUDA_ERR(hipGetLastError());
    CUDA_ERR(hipDeviceSynchronize());
    return output;
}

torch::Tensor launch_online_softmax_kernel_fp32(torch::Tensor x){
    CHECK_INPUT(x);    
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(x.device());
    const int N = x.numel();

    auto output = torch::zeros({N}, options);
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    online_softmax_kernel<256><<<grid, block>>>(x.data_ptr<float>(), output.data_ptr<float>(), N);
    CUDA_ERR(hipGetLastError());
    CUDA_ERR(hipDeviceSynchronize());
    return output;
}