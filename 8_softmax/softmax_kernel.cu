#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAException.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CUDA_ERR(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define WARP_SIZE 32
template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_shffl_sum(float val){
    // #params unroll
    for(int stride = kWarpSize >> 1; stride >= 1; stride >>= 1)
        val += __shfl_xor_sync(0xffffffff, val, stride);
    return val;
}

template<const int NUM_THREADS = 256>
__device__ __forceinline__ float block_reduce_sum_f32(float val){
    static __shared__ float shared[256];
    int lane = threadIdx.x % WARP_SIZE;
    int wid = threadIdx.x / WARP_SIZE;
    constexpr int warp_num = (NUM_THREADS - 1 + WARP_SIZE)/ WARP_SIZE;
    
    float t_val = warp_shffl_sum<WARP_SIZE>(val);
    if(lane == 0) shared[wid] = t_val;
    __syncthreads();

    t_val = (lane < warp_num) ? shared[lane] : 0.0f;
    t_val = warp_shffl_sum(t_val);
    t_val = __shfl_sync(0xffffffff, t_val, 0, 32);
    return t_val;
}



template<const int NUM_THREADS = 256>
__global__ void softmax_kernel(const float* x, float* y, float* total, int length){

    int local_tid = threadIdx.x;
    int global_tid = blockIdx.x * NUM_THREADS + threadIdx.x;

    float exp_val = global_tid < length ? expf(x[global_tid]) : 0.0f;
    float exp_sum = block_reduce_sum_f32<NUM_THREADS>(exp_val);

    if(local_tid == 0) atomicAdd(total, exp_sum);
    __threadfence();

    if(global_tid < length) y[global_tid] = exp_val / (*total);
}


torch::Tensor launch_softmax_kernel_fp32(torch::Tensor x){
    CHECK_INPUT(x);    
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(x.device());
    const int N = x.numel();

    auto output = torch::empty({N}, options);
    auto total = torch::zeros({1}, options);
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    softmax_kernel<256><<<grid, block>>>(x.data_ptr<float>(), output.data_ptr<float>(), total.data_ptr<float>(), N);
    CUDA_ERR(hipGetLastError());
    CUDA_ERR(hipDeviceSynchronize());
    return output;
}
